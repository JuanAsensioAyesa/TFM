#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf

// This is called by the host only
void cpu_kernel(const nanovdb::NanoGrid<float>* cpuGrid)
{
    printf("NanoVDB cpu; %4.2f\n", cpuGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the device only
__global__ void gpu_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
    printf("NanoVDB gpu: %4.2f\n", deviceGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the client code on the host
void launch_kernels(const nanovdb::NanoGrid<float>* deviceGrid,
                               const nanovdb::NanoGrid<float>* cpuGrid,
                               hipStream_t                    stream)
{
    gpu_kernel<<<1, 1, 0, stream>>>(deviceGrid); // Launch the device kernel asynchronously

    cpu_kernel(cpuGrid); // Launch the host "kernel" (synchronously)
}

// void scaleActiveVoxels(nanovdb::FloatGrid *grid_d, uint64_t leafCount, float scale)
// {
//     auto kernel = [grid_d, scale] __device__ (const uint64_t n) {
//         auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
//         const int i = n & 511;
//         const float v = scale * leaf_d->getValue(i);
//         if (leaf_d->isActive(i)) {
//             leaf_d->setValueOnly(i, v);// only possible execution divergence
//         }
//     };

//     thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
//     thrust::for_each(iter, iter + 512*leafCount, kernel);
// }

void createCubeCPU(nanovdb::GridBuilder<float>* cpuGrid){
    //nanovdb::NanoGrid<float>::accessor accessor = cpuGrid.getAccessor();
    nanovdb::Coord origin(0,0,0);
    int &i = origin[0], &j = origin[1], &k = origin[2];
    int size = 100;
    int depth = 100;
    int min_size = origin[0]-size;
    int min_depth = origin[1] - depth;
    int min_size_2 = origin[2]-size;
    int k_0 = origin[2];
    int i_0 = origin[0];
    int j_0 = origin[1];

    auto accessor = cpuGrid->getAccessor();
    
    int increment = 1;
    for(k=k_0 ;k>min_size_2;k-=increment){
        for(i=i_0;i > min_size;i-=increment){
            for(j=j_0 ;j>min_depth;j-=increment){
                accessor.setValue(origin,0.1);
                //std::cout<<origin<<std::endl;
            }
            //std::cout<<i<<std::endl;
        }
    }
}

// This is called by the client code on the host
void createCubes(const nanovdb::GridBuilder<float>* deviceGrid,
    const nanovdb::GridBuilder<float>* cpuGrid,
    hipStream_t                    stream)
{
    //dim3 threads_per_block(10, 10, 10);
    //dim3 number_of_blocks(13, 13, 14);

    //createCubeCPU(cpuGrid);
}

