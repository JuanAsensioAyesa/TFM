#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>

#include <nanovdb/util/Primitives.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include "pruebaThrust.h"

// This is called by the host only
void cpu_kernel(const nanovdb::NanoGrid<float>* cpuGrid)
{
    printf("NanoVDB cpu; %4.2f\n", cpuGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the device only
__global__ void gpu_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
    printf("NanoVDB gpu: %4.2f\n", deviceGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the client code on the host
void launch_kernels(const nanovdb::NanoGrid<float>* deviceGrid,
                               const nanovdb::NanoGrid<float>* cpuGrid,
                               hipStream_t                    stream)
{
    gpu_kernel<<<1, 1, 0, stream>>>(deviceGrid); // Launch the device kernel asynchronously

    cpu_kernel(cpuGrid); // Launch the host "kernel" (synchronously)
}

void scaleActiveVoxels(nanovdb::FloatGrid *grid_d, uint64_t leafCount, float scale)
{
    auto kernel = [grid_d, scale] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;
        const float v = scale * leaf_d->getValue(i);
        if (leaf_d->isActive(i)) {
            leaf_d->setValueOnly(i, v);// only possible execution divergence
        }
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void setZero(nanovdb::FloatGrid *grid_d,uint64_t leafCount){
    
    auto kernel = [grid_d] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;
        //const float v = scale * leaf_d->getValue(i);
        if (leaf_d->isActive(i)) {
            leaf_d->setValueOnly(i, 0);// only possible execution divergence
        }
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}




