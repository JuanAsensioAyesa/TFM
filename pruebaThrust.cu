#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>

#include <nanovdb/util/Primitives.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include "pruebaThrust.h"

// This is called by the host only
void cpu_kernel(const nanovdb::NanoGrid<float>* cpuGrid)
{
    printf("NanoVDB cpu; %4.2f\n", cpuGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the device only
__global__ void gpu_kernel(const nanovdb::NanoGrid<float>* deviceGrid)
{
    printf("NanoVDB gpu: %4.2f\n", deviceGrid->tree().getValue(nanovdb::Coord(99, 0, 0)));
}

// This is called by the client code on the host
void launch_kernels(const nanovdb::NanoGrid<float>* deviceGrid,
                               const nanovdb::NanoGrid<float>* cpuGrid,
                               hipStream_t                    stream)
{
    gpu_kernel<<<1, 1, 0, stream>>>(deviceGrid); // Launch the device kernel asynchronously

    cpu_kernel(cpuGrid); // Launch the host "kernel" (synchronously)
}

void scaleActiveVoxels(nanovdb::FloatGrid *grid_d, uint64_t leafCount, float scale)
{
    auto kernel = [grid_d, scale] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;
        const float v = scale * leaf_d->getValue(i);
        if (leaf_d->isActive(i)) {
            leaf_d->setValueOnly(i, v);// only possible execution divergence
        }
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void setZero(nanovdb::FloatGrid *grid_d,uint64_t leafCount){
    
    auto kernel = [grid_d] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;
        float scale = 2;
        const float v = scale * leaf_d->getValue(i);
        if (leaf_d->isActive(i)) {
            leaf_d->setValueOnly(i, 0);// only possible execution divergence
        }
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

inline float averageSurrounding(nanovdb::Coord coordenadas,nanovdb::FloatGrid *grid_source,nanovdb::FloatGrid *grid_destiny){
    float incrementos[] = {-1,0,1};
    int len_incrementos = 3;
    float accum = 0.0;
    nanovdb::Coord new_coord;
    float new_vec[3];
    auto vec = coordenadas.asVec3s();
    for(int i_incremento_x = 0;i_incremento_x<len_incrementos;i_incremento_x++){
        for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos;i_incremento_y++){
            for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos;i_incremento_z++){
                int incremento_x = incrementos[i_incremento_x];
                int incremento_y = incrementos[i_incremento_y];
                int incremento_z = incrementos[i_incremento_z];

                
                new_vec[0] = vec[0]+incremento_x;
                new_vec[1] = vec[1]+incremento_y;
                new_vec[2] = vec[2]+incremento_z;

                new_coord = nanovdb::Coord(new_vec[0],new_vec[1],new_vec[2]);
                accum += grid_source->tree().getValue(new_coord);
                
            }
        }
    }
    //std::cout<<vec<<std::endl;
    accum = accum /(len_incrementos * len_incrementos * len_incrementos);

    
    return accum;
}

void average(nanovdb::FloatGrid *grid_source,nanovdb::FloatGrid *grid_destiny,uint64_t leafCount){
    auto kernel = [grid_source,grid_destiny] __device__ (const uint64_t n) {
        auto *leaf_d = grid_source->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;//Es el tamanio por defecto de leafNodes
        auto *leaf_d_destiny = grid_destiny->tree().getFirstNode<0>() + (n >> 9);
        if (leaf_d->isActive(i)) {
            auto coord = leaf_d->offsetToGlobalCoord(i);
            float incrementos[] = {-1,0,1};
            int len_incrementos = 3;
            float accum = 0.0;
            nanovdb::Coord new_coord = nanovdb::Coord();
            //float new_vec[3];
            auto vec = coord.asVec3s();
            
            //printf("%d %f %f %f\n",i,vec[0],coord[1],vec[2]);
            auto acc = grid_source->tree().getAccessor();
            auto acc_destiny = grid_destiny->tree().getAccessor();
            int incrementadas = 0 ;
            for(int i_incremento_x = 0;i_incremento_x<len_incrementos;i_incremento_x++){
                for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos;i_incremento_y++){
                    for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos;i_incremento_z++){
                        int incremento_x = incrementos[i_incremento_x];
                        int incremento_y = incrementos[i_incremento_y];
                        int incremento_z = incrementos[i_incremento_z];

                        
                        // new_vec[0] = vec[0]+incremento_x;
                        // new_vec[1] = vec[1]+incremento_y;
                        // new_vec[2] = vec[2]+incremento_z;

                        if(acc.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
                            float aux = acc.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            accum += aux;
                            incrementadas++;
                        }
                        
                        
                    }
                }
            }
            //std::cout<<vec<<std::endl;
            accum = accum /incrementadas;
            //printf("%f \n",accum);
            //printf("%f\n",accum);
            //acc.setValue(coord,accum);
            leaf_d_destiny->setValueOnly(i, accum);// only possible execution divergence
        }
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}



