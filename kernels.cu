#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>

#include <nanovdb/util/Primitives.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include "pruebaThrust.h"
#include <nanovdb/util/Stencils.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


const float threshold_vecino = 0;
const float time_factor = 6*60; //Timestep de 6 minutos pasado a segundos
__device__ const float ini_endothelial = 1.0;

/**
 * @brief Genera la estrucura inicial de las celulas endoteliales, cada cilindro tendra el tamanio de un leaf node (8x8x8)
 * 
 */
void generateEndothelial(nanovdb::FloatGrid *grid_d, uint64_t leafCount, int lim_sup,int lim_inf,int modulo)
{
    auto kernel = [grid_d, lim_sup,lim_inf,modulo] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;
        
        auto coord_indi = leaf_d->offsetToGlobalCoord(i);
        auto coord = leaf_d->origin();
        coord = coord_indi;
        //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
        if(coord[1]>lim_inf && coord[1]<lim_sup){
            if(coord[0]%modulo == 0 && coord[2]%modulo == 0 ){
                leaf_d->setValueOnly(i,ini_endothelial);
            }
        }
        if(coord_indi[1]==lim_inf || coord_indi[1]==lim_sup){
            leaf_d->setValueOnly(i,ini_endothelial);
        }
        
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

/**
 * @brief Implementa la ecuacion en diferencias relacionada con el TAF 
 *  (Ecuacion 8)
 * 
 */
void equationTAF(nanovdb::FloatGrid* input_grid_endothelial,nanovdb::FloatGrid* input_grid_TAF,nanovdb::FloatGrid* output_grid_TAF,uint64_t leafCount){
    auto kernel = [input_grid_endothelial,input_grid_TAF,output_grid_TAF] __device__ (const uint64_t n){
        auto *leaf_d = output_grid_TAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = input_grid_TAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        //Coordenadas del voxel globales
        auto coord = leaf_d->offsetToGlobalCoord(i);
        auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
        auto accessor_TAF_in = input_grid_TAF->tree().getAccessor();
        auto accessor_TAF_out = output_grid_TAF->tree().getAccessor();
        int incrementos_vecinos[] = {-1,0,1};
        int len_incrementos = 3;
        
        float n_i = 0;
        bool esVecino = false;
        //Se calcula n_i , que determina si se es vecino de una endothelial
        //if(accessor_endothelial.getValue(coord)<threshold_vecino){
            for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
                for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
                    for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
                        int incremento_x = incrementos_vecinos[i_incremento_x];
                        int incremento_y = incrementos_vecinos[i_incremento_y];
                        int incremento_z = incrementos_vecinos[i_incremento_z];
    
    
                        if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
                            n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            
                        }
                        esVecino = n_i > threshold_vecino;//Esto igual esta feo
                        
                    }
                }
            }
        //}

        float n_c = 0.025;
        //printf("%f\n",n_i);
        if(esVecino){
            n_i = 1.0;
        }else{
            n_i  = 0.0;
        }
        float c = leaf_s->getValue(i);
        
        
        float derivative = -n_c * n_i* c;
        // if(c > 0){
        //     printf("%f %f %f\n",c,n_i,derivative);
        // }
        float old_c = leaf_s->getValue(i);
        auto new_value = old_c + derivative * time_factor;
        if(new_value < 0 ){
            new_value = 0 ;
        }
        if(new_value >1 ){
            new_value = 1;
        }
        leaf_d->setValueOnly(i,new_value);
        

        

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationFibronectin(nanovdb::FloatGrid* input_grid_endothelial,nanovdb::FloatGrid* input_grid_Fibronectin,nanovdb::FloatGrid* input_grid_MDE,nanovdb::FloatGrid* output_grid_Fibronectin,uint64_t leafCount){
    auto kernel = [input_grid_endothelial,input_grid_Fibronectin,input_grid_MDE,output_grid_Fibronectin] __device__ (const uint64_t n) {
        auto *leaf_d = output_grid_Fibronectin->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = input_grid_Fibronectin->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_mde = input_grid_MDE->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        //Coordenadas del voxel globales
        auto coord = leaf_d->offsetToGlobalCoord(i);
        auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
        auto accessor_Fibronectin_in = input_grid_Fibronectin->tree().getAccessor();
        auto accessor_Fibronectin_out = output_grid_Fibronectin->tree().getAccessor();
        int incrementos_vecinos[] = {-1,0,1};
        int len_incrementos = 3;
        
        float n_i = 0;
        bool esVecino = false;
        //Se calcula n_i , que determina si se es vecino de una endothelial
        //if(accessor_endothelial.getValue(coord)<threshold_vecino){
            for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
                for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
                    for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
                        int incremento_x = incrementos_vecinos[i_incremento_x];
                        int incremento_y = incrementos_vecinos[i_incremento_y];
                        int incremento_z = incrementos_vecinos[i_incremento_z];
    
    
                        if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
                            n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            
                        }
                        esVecino = n_i > threshold_vecino;//Esto igual esta feo
                        
                    }
                }
            }
        //}
        
        
        float production_rate = 0.0125;
        float degradation_rate = 0.1;
        if(esVecino){
            n_i = 1.0;
        }else{
            n_i = 0.0;
        }
        
        float old_f = leaf_s->getValue(i);
        float old_mde = leaf_mde->getValue(i);

        float derivative = production_rate * n_i - degradation_rate * old_f * old_mde;
        // if(derivative > 0 ){
        //     printf("%f\n",old_f + derivative * time_factor);
        // }
        auto new_value = old_f + derivative*time_factor;
        if(new_value < 0 ){
            new_value =  0;
        }
        if(new_value > 1){
            new_value = 1;
        }
        leaf_d->setValueOnly(i,new_value);
        //leaf_d->setValueOnly(i,n_i);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationMDE(nanovdb::FloatGrid* input_grid_endothelial,nanovdb::FloatGrid* input_grid_MDE,nanovdb::FloatGrid* output_grid_MDE,uint64_t leafCount){
    auto kernel = [input_grid_endothelial,input_grid_MDE,output_grid_MDE] __device__ (const uint64_t n) {
        auto *leaf_d = output_grid_MDE->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = input_grid_MDE->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        //Coordenadas del voxel globales
        auto coord = leaf_d->offsetToGlobalCoord(i);
        auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
        auto accessor_Fibronectin_in = input_grid_MDE->tree().getAccessor();
        auto accessor_Fibronectin_out = output_grid_MDE->tree().getAccessor();
        int incrementos_vecinos[] = {-1,0,1};
        int len_incrementos = 3;
        
        float n_i = 0;
        bool esVecino = false;
        //Se calcula n_i , que determina si se es vecino de una endothelial
        //if(accessor_endothelial.getValue(coord)<threshold_vecino){
            for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
                for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
                    for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
                        int incremento_x = incrementos_vecinos[i_incremento_x];
                        int incremento_y = incrementos_vecinos[i_incremento_y];
                        int incremento_z = incrementos_vecinos[i_incremento_z];
    
    
                        if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
                            n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            
                        }
                        esVecino = n_i > threshold_vecino;//Esto igual esta feo
                        
                    }
                }
            }
        //}
        float production_rate = 0.0000015;
        
        float diffussion_coefficient = 0.0025;
        float degradation_rate = 0.75;
        //printf("%f %f %f\n",production_rate,diffussion_coefficient,degradation_rate);
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*input_grid_MDE);
        stencilNano.moveTo(coord);
        float laplacian = stencilNano.laplacian();
        if(esVecino){
            n_i = 1.0;
        }else{
            n_i = 0.0;
        }
        float old_mde = leaf_s->getValue(i);
        float derivative = n_i * production_rate + diffussion_coefficient * laplacian * old_mde - degradation_rate * old_mde;
        // float factor_1 = n_i * production_rate;
        // float factor_2 = diffussion_coefficient * laplacian * old_mde;
        // float factor_3 = degradation_rate * old_mde;
        // if(laplacian!=0){
        //     printf("%f\n",laplacian);
        // }
        
        // if(factor_1 != 0|| factor_2 != 0 || factor_3!=0){
        //     printf("%f %f %f\n",n_i * production_rate,diffussion_coefficient*laplacian*old_mde,degradation_rate*old_mde);

        // }
        // if(derivative > 0 && factor_1 > 0.000002){
        //     printf("%f %f %f\n",factor_1,factor_2,factor_3);
        // }
        //float derivative = diffussion_coefficient * laplacian;
        auto new_value = old_mde + derivative * time_factor;
        if(new_value < 0 ){
            new_value = 0;
        }
        if(new_value >1 ){
            new_value = 1;
        }
        leaf_d->setValueOnly(i,n_i);
        //leaf_d->setValueOnly(i,n_i);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void pruebaGradiente(nanovdb::Vec3fGrid  *grid_d,nanovdb::FloatGrid* gridSource ,uint64_t leafCount)
{
    auto kernel = [grid_d,gridSource] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = gridSource->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf_d->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridSource);
        
        auto accessor = gridSource->tree().getAccessor();
        
        stencilNano.moveTo(coord_nano);
        auto gradiente = stencilNano.gradient();
        // s
        // auto suma = gradiente[0]+gradiente[1]+gradiente[2];
        // if(suma!=0){
        //     nanovdb::Coord coord_aux = coord;
        //     coord_aux[0] = 0 ;
        //     coord_aux[1] = 0 ;
        //     coord_aux[2] = 0 ;
        //     //printf("%f %f\n",suma,leaf_s->getValue(coord));
        //     printf("%f \n",accessor.getValue(coord));
        // }
        leaf_d->setValueOnly(coord,gradiente);
        
        
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}


__device__ float chemotacticSensivity(float c){
    float chemotacticMigration = 0.38;
    float chemotacticConstant = 0.6;
    return chemotacticMigration /(1 + chemotacticConstant*c);
}
/*
    Ecuacion 6
*/
void equationEndothelial(nanovdb::FloatGrid * grid_s,nanovdb::FloatGrid * grid_d,nanovdb::FloatGrid* gridTAF,nanovdb::FloatGrid* gridFibronectin,nanovdb::Vec3fGrid* gradientTAF,nanovdb::Vec3fGrid* gradientFibronectin,uint64_t leafCount){
    auto kernel = [grid_s,grid_d,gridTAF,gridFibronectin,gradientTAF,gradientFibronectin] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = grid_s->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_taf = gridTAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf_d->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        /*
            Primera parte: Difusion aleatoria
        */
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*grid_s);
        stencilNano.moveTo(coord_nano);
        float old_n = leaf_s->getValue(coord_nano);
        float laplacian = stencilNano.laplacian();
        //printf("%f\n",laplacian);
        float factorEndothelial = laplacian * 0.0003 ;
        /*
            Segunda parte, chimiotaxis TAF
        */
        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencilTAF(*gradientTAF);
        
        stencilTAF.moveTo(coord_nano);
        auto gradientTAF = stencilTAF.gradient();
        float taf_value = leaf_taf->getValue(coord_nano);
        // for(int index = 0 ;index <3;index++){
        //     gradientTAF[index] *= chemotacticSensivity(taf_value);
        // }
        float factorTAF = gradientTAF[0][0] + gradientTAF[1][1] + gradientTAF[2][2];
        //factorTAF *=10;
        //printf("%f  %f  %f\n",gradientTAF[0][0],gradientTAF[1][1],gradientTAF[2][2]);
        
        //float factorTAF  = stencilNano.gaussianCurvature() ;
        /*
            Tercera parte, Fibronectin
        */
        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencilFibronectin(*gradientFibronectin);
        stencilFibronectin.moveTo(coord_nano);
        auto gradientFibronectin = stencilFibronectin.gradient();
        float factorFibronectin = gradientFibronectin[0][0] + gradientFibronectin[1][1] + gradientFibronectin[2][2];
        factorFibronectin = factorFibronectin * 0.28;
        
        //printf("%f %f %f\n",factorEndothelial,factorTAF,factorFibronectin);

        //float derivative = factorEndothelial  + factorTAF ;//+ factorFibronectin;
        float derivative = factorEndothelial - factorTAF - factorFibronectin;
        // if(derivative > 0 ){
        //     printf("%f\n",derivative);
        // }
        // if(derivative > 100){
        //     printf("%f %f %f\n",factorEndothelial,factorTAF,factorFibronectin);
        // }
        //float derivative = -factorTAF;
        auto new_value = old_n + derivative * time_factor;
        if(new_value < 0 ){
            new_value = 0 ;
        }
        if(new_value > 1){
            new_value = 1;
        }
        leaf_d->setValueOnly(coord_nano,new_value);//6 minutos //* 60 segundos
        //leaf_d->setValueOnly(coord_nano,derivative);//6 minutos //* 60 segundos

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
__device__ float computeEndothelial(nanovdb::Coord coord_nano,nanovdb::CurvatureStencil<nanovdb::FloatGrid>& stencilEndothelial,nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> &stencilTAF,nanovdb::CurvatureStencil<nanovdb::Vec3fGrid>& stencilFibronectin){
    /*
    Primera parte: Difusion aleatoria
    */
    
    stencilEndothelial.moveTo(coord_nano);
    //float old_n = leaf_s->getValue(coord_nano);
    float laplacian = stencilEndothelial.laplacian();
    //printf("%f\n",laplacian);
    float factorEndothelial = laplacian * 0.0003 ;

    /*
    Segunda parte, chimiotaxis TAF
    */
    
    
    stencilTAF.moveTo(coord_nano);
    auto gradientTAF = stencilTAF.gradient();
    float factorTAF = gradientTAF[0][0] + gradientTAF[1][1] + gradientTAF[2][2];
    /*
    Tercera parte, Fibronectin
    */
    
    stencilFibronectin.moveTo(coord_nano);
    auto gradientFibronectin = stencilFibronectin.gradient();
    float factorFibronectin = gradientFibronectin[0][0] + gradientFibronectin[1][1] + gradientFibronectin[2][2];
    factorFibronectin = factorFibronectin * 0.28;
    
    //printf("%f %f\n",factorTAF,factorFibronectin);

    float derivative = factorEndothelial  - factorTAF - factorFibronectin;
    //derivative = -factorTAF-factorFibronectin;
    derivative = -factorTAF;
    return derivative;
}
__device__ bool  isNextToEndothelial(nanovdb::Coord coord,nanovdb::FloatGrid * input_grid_endothelial){
    auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
    int incrementos_vecinos[] = {-1,0,1};
    int len_incrementos = 3;
    float n_i = 0;
    bool esVecino = false;
    //Se calcula n_i , que determina si se es vecino de una endothelial
    //if(accessor_endothelial.getValue(coord)<threshold_vecino){
    //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
    for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
        for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
            for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
                int incremento_x = incrementos_vecinos[i_incremento_x];
                int incremento_y = incrementos_vecinos[i_incremento_y];
                int incremento_z = incrementos_vecinos[i_incremento_z];


                if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
                    n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                    esVecino = n_i > threshold_vecino;//Esto igual esta feo
                }else{
                    esVecino = false;
                }
                
                
            }
        }
    }
    return esVecino;
}
__device__ bool isNextToEndothelialDiscrete(nanovdb::Coord coord,nanovdb::FloatGrid * input_grid_endothelial){
    auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
    int desplazamientos[] = {-1,1};
    int len_desp = 2;
    
    //int desplazamiento_max[3];
    
    
    //nanovdb::Coord coord_max = coord;
    //float max_derivative = computeEndothelial(coord,stencilEndothelial,stencilTAF,stencilFibronectin);

    //Se calcula el maximo
    bool esVecino = false;
    for(int dimension = 0 ;dimension <3&&!esVecino;dimension++){

        for(int desplazamiento = 0;desplazamiento<len_desp&&!esVecino;desplazamiento++){
            nanovdb::Coord new_coord = coord;
            new_coord[dimension] += desplazamientos[desplazamiento];
            float value_i = accessor_endothelial.getValue(new_coord);
            esVecino = value_i > threshold_vecino;
        }
    }
    return esVecino;
}
__device__ int getPosition(nanovdb::Coord coord_self,nanovdb::FloatGrid * endothelialTip){
    auto accessor = endothelialTip->tree().getAccessor();
    nanovdb::Coord coord_i = coord_self;

    coord_i[2] = coord_i[2] - 1; //Comprobamos frente
    if(accessor.getValue(coord_i) != 0 ){
        return 6;//coord_self esta a la dch de la celula endothelial
    }
    coord_i[2] = coord_self[2];

    coord_i[2] = coord_i[2] + 1; //Comprobamos frente
    if(accessor.getValue(coord_i) != 0 ){
        return 5;//coord_self esta a la dch de la celula endothelial
    }
    coord_i[2] = coord_self[2];

    coord_i[1] = coord_i[1] +1; //Comprobamos arriba;
    if(accessor.getValue(coord_i) != 0 ){
        return 3;//coord_self esta debajo de la celula endothelial
    }
    coord_i[1] = coord_self[1];

    coord_i[0] = coord_i[0] +1; //Comprobamos derecha
    if(accessor.getValue(coord_i) != 0 ){
        return 4;//coord_self esta a la izq de la celula endothelial
    }
    coord_i[0] = coord_self[0];

    coord_i[1] = coord_i[1] - 1; //Comprobamos abajo
    if(accessor.getValue(coord_i) != 0 ){
        return 1;//coord_self esta arriba de la celula endotelial
    }
    coord_i[1] = coord_self[1];

    coord_i[0] = coord_i[0] - 1; //Comprobamos izquierda
    if(accessor.getValue(coord_i) != 0 ){
        return 2;//coord_self esta a la dch de la celula endothelial
    }
    coord_i[0] = coord_self[0];

    
    return 0;


}

__device__ bool isMax(nanovdb::Coord coord_self,nanovdb::FloatGrid * endothelialTip,nanovdb::FloatGrid * endothelial,nanovdb::FloatGrid* endothelialDiscrete){
    auto accessor_endothelial = endothelial->tree().getAccessor();
    auto accessor_discrete = endothelialDiscrete->tree().getAccessor();
    int positionSelf = getPosition(coord_self,endothelialTip);
    //printf("%d\n",positionSelf);
    nanovdb::Coord coord_endothelial = coord_self;
    switch(positionSelf){
        case 1:
            coord_endothelial[1] -=1;
            break;
        case 2:
            coord_endothelial[0] -= 1;
            break;
        case 3:
            coord_endothelial[1] += 1;
            break;
        case 4:
            coord_endothelial[0] += 1;
            break;
        case 5:
            coord_endothelial[2] += 1;
            break;
        case 6:
            coord_endothelial[2] -= 1;
            break;
        default:
            break;

    };
    int desplazamientos[] = {-1,1};
    int len_desp = 2;
    
    nanovdb::Coord coord_max;
    float value_max = -1;
    if(positionSelf != 0 ){
        for(int dimension = 0 ;dimension <3;dimension++){

            for(int desplazamiento = 0;desplazamiento<len_desp;desplazamiento++){
                nanovdb::Coord new_coord = coord_endothelial;
                new_coord[dimension] += desplazamientos[desplazamiento];
                float value_i = accessor_endothelial.getValue(new_coord);
                //printf("Position Self %d, new_coord %d %d %d value_i %f\n",positionSelf,new_coord[0],new_coord[1],new_coord[2],value_i);
                if(accessor_discrete.getValue(new_coord) == 0 && value_i > value_max){
                    value_max = value_i;
                    coord_max = new_coord;
                }
            }
        }
    }
    //return positionSelf == 4;
    return positionSelf != 0 && coord_max == coord_self;


}

__device__ void moveRandom(nanovdb::Coord coord_self,nanovdb::FloatGrid* gridEndothelial,nanovdb::FloatGrid* gridEndothelialDiscrete,nanovdb::FloatGrid* gridTip,float randomValue,int n ){
    auto accessor_endothelial = gridEndothelial->tree().getAccessor();
    auto accessor_discrete = gridEndothelialDiscrete->tree().getAccessor();
    auto accessor_tip = gridTip->tree().getAccessor();
    auto *leaf_tip = gridTip->tree().getFirstNode<0>()+(n>>9);
    auto *leaf_discrete = gridEndothelialDiscrete->tree().getFirstNode<0>()+(n>>9);
    int positionSelf = getPosition(coord_self,gridTip);
    //printf("%d\n",positionSelf);
    nanovdb::Coord coord_endothelial = coord_self;
    switch(positionSelf){
        case 1:
            coord_endothelial[1] -=1;
            break;
        case 2:
            coord_endothelial[0] -= 1;
            break;
        case 3:
            coord_endothelial[1] += 1;
            break;
        case 4:
            coord_endothelial[0] += 1;
            break;
        case 5:
            coord_endothelial[2] += 1;
            break;
        case 6:
            coord_endothelial[2] -= 1;
            break;
        default:
            break;

    };
    if(positionSelf == 0 ){
        return;
    }


    int desplazamientos[] = {-1,1};
    const int len_desp = 2;
    
    nanovdb::Coord coord_max;
    float value_accum = 0.0;
    const int length = 3 * len_desp;
    float values[length];
    nanovdb::Coord coords[length];
    int i_value = 0;
    if(positionSelf != 0 ){
        for(int dimension = 0 ;dimension <3;dimension++){

            for(int desplazamiento = 0;desplazamiento<len_desp;desplazamiento++){
                nanovdb::Coord new_coord = coord_endothelial;
                new_coord[dimension] += desplazamientos[desplazamiento];
                float value_i = accessor_endothelial.getValue(new_coord);
                //printf("Position Self %d, new_coord %d %d %d value_i %f\n",positionSelf,new_coord[0],new_coord[1],new_coord[2],value_i);
                value_accum += value_i;
                values[i_value] = value_i;
                coords[i_value] = new_coord;
                i_value++;
            }
        }
    }
    // for(int i = 0 ;i<length;i++){
    //     values[i] = 1-values[i] / value_accum;
    // } 
    thrust::sort(thrust::device, values, values + length);
    bool decided = false;
    for(int i = 0 ;i<length && !decided;i++){
        if(coords[i]==coord_self && randomValue>values[i]){
            leaf_tip->setValue(coords[i],2);
            leaf_discrete->setValue(coords[i],1);
            decided = true;
        }
    }


}

void equationEndothelialDiscrete(nanovdb::FloatGrid * grid_source_discrete,nanovdb::FloatGrid * grid_destiny_discrete,nanovdb::FloatGrid* gridDerivativeEndothelial,nanovdb::FloatGrid* gridTAF,nanovdb::FloatGrid * gridTipRead,nanovdb::FloatGrid* gridTipWrite,int seed,uint64_t leafCount){
    thrust::minstd_rand rng;
    thrust::default_random_engine randEng;
    thrust::uniform_real_distribution<float> uniDist;
    int discard = seed;
    randEng.discard(discard);
    float random = uniDist(randEng);
    auto kernel = [grid_source_discrete,grid_destiny_discrete,gridDerivativeEndothelial,gridTAF,gridTipRead,gridTipWrite,rng,seed,random] __device__ (const uint64_t n) {
        auto *leaf_d = grid_destiny_discrete->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = grid_source_discrete->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_tip_write = gridTipWrite->tree().getFirstNode<0>()+(n>>9);
        auto *leaf_tip_read = gridTipRead->tree().getFirstNode<0>()+(n>>9);
        //auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        auto coord = leaf_tip_write->offsetToGlobalCoord(i);
        auto coord_d = leaf_d->offsetToGlobalCoord(i);
        float vector_probabilidades[] = {0.04,0.06,0.08,0.2};
        //float taf_value = leaf_TAF->getValue(coord);

        // if(coord[0] == 0&&coord[1]==0 && coord[2]==0){
        //     printf("RAndom %f\n",random);

        // }        

        //leaf_d->setValue(coord,random);

        // nanovdb::Coord coord_dummy;
        // coord_dummy[0] = 0 ;
        // coord_dummy[1] =0 ;
        // coord_dummy[2] = 0 ;
        //leaf_d->setValueOnly(coord_dummy,100);
        // if(leaf_tip_read->getValue(i)>0){
        //     printf("TIP\n");
        // }
       // static int first = true;
       if(isNextToEndothelialDiscrete(coord_d,gridTipRead)){
        //if(leaf_tip->getValue(i)>0){
            //if((coord_d[1]-1)%2 == 0){
            //if(coord_d[1]%2 == 0 ){
            int positionSelf = getPosition(coord_d,gridTipRead);
            //moveRandom(coord,gridDerivativeEndothelial,grid_destiny_discrete,gridTipWrite,random,n);
            if(isMax(coord_d,gridTipRead,gridDerivativeEndothelial,grid_source_discrete)){
                //printf("Is max %d\n",positionSelf);
                leaf_d->setValue(coord_d,1.0);
                leaf_tip_write->setValue(coord_d,2.0);
            
            }else{
                //leaf_tip_write->setValue(coord_d,0);
                
            }
            // }
            // leaf_d->setValue(coord_d,1.0);
            // //leaf_s->setValue(coord_d,1.0);
            // if(leaf_s->getValue(coord_d)==0){
            //     leaf_tip_write->setValue(coord_d,2.0);
            // }
            //leaf_tip->setValue(coord_d,0);
            //coord_d[0]+=1;
            //leaf_tip->setValue(coord_d,1);

            //FALTA EL BRANCHING
           
        }else{
            float value = leaf_s->getValue(i);
            leaf_d->setValue(coord_d,value);
        }
        // }else if(false&&isNextToEndothelialDiscrete(coord_d,grid_source_discrete)){
        //     //first = false;
        //     if(taf_value >= 0.8 && random >= 1.0-vector_probabilidades[3]){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else if(taf_value >=0.7 && random >= 1-vector_probabilidades[2] ){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else if(taf_value >= 0.5&& random >= 1-vector_probabilidades[1]){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else if(taf_value >=0.3&& random >= 1-vector_probabilidades[0]){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else{
        //         //NO hay branch
        //         //leaf_tip->setValueOnly(coord,0.0);
        //     }
        //     //leaf_d->setValueOnly(coord,1.0);
        // }else{
        //     leaf_tip_write->setValue(coord_d,0);
        // }
        
       
        //leaf_d->setValueOnly(i,0.0);
        
        
    
    
    
    
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void branching(nanovdb::FloatGrid* gridEndothelialTip,int leafCount){
    auto kernel = [gridEndothelialTip] __device__ (const uint64_t n) {
        auto* leaf_tip = gridEndothelialTip->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        auto coord_d = leaf_tip->offsetToGlobalCoord(i);
        float value = leaf_tip->getValue(i);
        float new_value = 1;
        if(true || i%2==0){
            new_value = value-1;
        }
        //float new_value = value-1;
        if(new_value < 0 ){
            new_value =0;
        }
        leaf_tip->setValue(coord_d,new_value);

        
        

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}


/*
    Genera el gradiente escalado del TAF, para poder calcular la divergencia
*/
void generateGradientTAF(nanovdb::FloatGrid * gridTAF,nanovdb::FloatGrid * gridTAFEndothelial,nanovdb::Vec3fGrid* gradientTAF,uint64_t leafCount){
    auto kernel = [gridTAF,gridTAFEndothelial,gradientTAF] __device__ (const uint64_t n) {
        auto *leaf_s = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAFEndothelial = gridTAFEndothelial->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Gradient = gradientTAF->tree().getFirstNode<0>() + (n >> 9);
        auto accessor_aux = gradientTAF->getAccessor();
        const int i = n & 511;
        auto coord = leaf_s->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridTAFEndothelial);
        //printf("%d %d %d\n",coord_nano[0],coord_nano[1],coord_nano[2]);
        stencilNano.moveTo(coord_nano);
        auto gradient = stencilNano.gradient();
        float sensivity = chemotacticSensivity(leaf_s->getValue(i));
        
        gradient = gradient *sensivity;
        // if(coord[0]== 0 || coord[1]==0||coord[2]==0){
        //     gradient[0] = 0;
        //     gradient[1] = 0 ;
        //     gradient[2] = 0;
        // }
        // if(gradient[0]!=0){
        //     printf("%f %f\n",sensivity,endothelialValue);
        // }
        
        
        //gradient[0] = 2.0;
        //gradient[1] = 1.0 ;
        //gradient[2] = 3.0;
        leaf_Gradient->setValueOnly(i,gradient);
        //auto aux = accessor_aux.getValue(coord);
        //printf("%f %f %f\n",aux[0],aux[1],aux[2]);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

/*
    Genera el gradiente escalado de la Fibronectina, para poder calcular la divergencia
*/
void generateGradientFibronectin(nanovdb::FloatGrid * gridFibronectin,nanovdb::FloatGrid * gridEndothelial,nanovdb::Vec3fGrid* gradientFibronectin,uint64_t leafCount){
    auto kernel = [gridFibronectin,gridEndothelial,gradientFibronectin] __device__ (const uint64_t n) {
        auto *leaf_s = gridFibronectin->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Gradient = gradientFibronectin->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        auto coord = leaf_s->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridFibronectin);
        stencilNano.moveTo(coord_nano);
        auto gradient = stencilNano.gradient();
        
        float endothelialValue = leaf_Endothelial->getValue(i);
        gradient = gradient  * endothelialValue;
        
        leaf_Gradient->setValueOnly(coord,gradient);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void divergence(nanovdb::Vec3fGrid *grid_s,nanovdb::FloatGrid *grid_d,uint64_t leafCount){
    auto kernel = [grid_s,grid_d] __device__ (const uint64_t n) {
        auto *leaf_s = grid_s->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        auto coord = leaf_s->offsetToGlobalCoord(i);

        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencil(*grid_s);
        stencil.moveTo(coord);
        auto gradient = stencil.gradient();
        auto divergence = gradient[0][0]+gradient[1][1] + gradient[2][2];
        leaf_d->setValueOnly(coord,divergence);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
void laplacian(nanovdb::FloatGrid * grid_s,nanovdb::FloatGrid * grid_d, uint64_t leafCount){
    auto kernel = [grid_s,grid_d] __device__ (const uint64_t n) {
        auto *leaf_s = grid_s->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        auto coord = leaf_s->offsetToGlobalCoord(i);

        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencil(*grid_s);
        const nanovdb::Coord coord_nano = coord;
        stencil.moveTo(coord_nano);
        auto old_value = leaf_s->getValue(i);
        auto laplacian = stencil.laplacian();
        // if(laplacian!= 0){
        //     printf("%f\n",laplacian);
        // }
        auto new_value = old_value + laplacian*0.1;

        // if(new_value < 0 ){
        //     new_value = 0;
        // }
        
        leaf_d->setValueOnly(i,new_value);
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void product(nanovdb::FloatGrid * gridTAF,nanovdb::FloatGrid * gridEndothelial,nanovdb::FloatGrid *grid_d, uint64_t leafCount){
    auto kernel = [gridTAF,gridEndothelial,grid_d] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        //auto coord = leaf_d->offsetToGlobalCoord(i);

        auto new_value = leaf_TAF->getValue(i)*leaf_Endothelial->getValue(i);
        //new_value = leaf_TAF->getValue(i);
        leaf_d->setValueOnly(i,new_value);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void cleanEndothelial(nanovdb::FloatGrid * gridEndothelial,uint64_t leafCount){
    auto kernel = [gridEndothelial] __device__ (const uint64_t n) {
        auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;

        auto coord = leaf_Endothelial->offsetToGlobalCoord(i);
        // if(coord[1]<-149){
        //     printf("%d\n",coord[1]);
        // }
        if(coord[0] == -250 || coord[1] == -150 || coord[2] == -250){
            leaf_Endothelial->setValueOnly(i,0.00001);
            leaf_Endothelial->setValueOnly(i,0);
            //printf("Uese\n");
        }
        if(coord[0] == 0 || coord[1] == 0 || coord[2] == 0){
            leaf_Endothelial->setValueOnly(i,0.00001);
            leaf_Endothelial->setValueOnly(i,0);
            //printf("Uese\n");
        }
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}