#include "hip/hip_runtime.h"
// Copyright Contributors to the OpenVDB Project
// SPDX-License-Identifier: MPL-2.0
#include <nanovdb/util/GridBuilder.h>
#include <nanovdb/NanoVDB.h> // this defined the core tree data structure of NanoVDB accessable on both the host and device
#include <stdio.h> // for printf
#include <thrust/iterator/counting_iterator.h>
#include <thrust/for_each.h>

#include <nanovdb/util/Primitives.h>
#include <nanovdb/util/CudaDeviceBuffer.h>
#include "pruebaThrust.h"
#include <nanovdb/util/Stencils.h>
#include <thrust/random/uniform_real_distribution.h>
#include <thrust/random/linear_congruential_engine.h>
#include <thrust/device_vector.h>
#include <thrust/random.h>
#include <thrust/sort.h>
#include <thrust/execution_policy.h>


const float threshold_vecino = 0;
const float time_factor = 6; //Timestep de 6 minutos pasado a segundos
__device__ const float ini_endothelial = 1.0;

/**
 * @brief Genera la estrucura inicial de las celulas endoteliales, cada cilindro tendra el tamanio de un leaf node (8x8x8)
 * 
 */
void generateEndothelial(nanovdb::FloatGrid *grid_d, uint64_t leafCount, int lim_sup,int lim_inf,int modulo)
{
    auto kernel = [grid_d, lim_sup,lim_inf,modulo] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        const int i = n & 511;
        
        auto coord_indi = leaf_d->offsetToGlobalCoord(i);
        auto coord = leaf_d->origin();
        coord = coord_indi;
        //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
        if(coord[1]>lim_inf && coord[1]<lim_sup){
            if(coord[0]%modulo == 0 && coord[2]%modulo == 0 ){
                leaf_d->setValueOnly(i,ini_endothelial);
            }
        }
        if(coord_indi[1]==lim_inf || coord_indi[1]==lim_sup){
            leaf_d->setValueOnly(i,ini_endothelial);
        }
        
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

/**
 * @brief Implementa la ecuacion en diferencias relacionada con el TAF 
 *  (Ecuacion 8)
 * 
 */
void equationTAF(nanovdb::FloatGrid* input_grid_endothelial,nanovdb::FloatGrid* input_grid_TAF,nanovdb::FloatGrid* output_grid_TAF,uint64_t leafCount){
    auto kernel = [input_grid_endothelial,input_grid_TAF,output_grid_TAF] __device__ (const uint64_t n){
        auto *leaf_d = output_grid_TAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = input_grid_TAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        //Coordenadas del voxel globales
        auto coord = leaf_d->offsetToGlobalCoord(i);
        auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
        auto accessor_TAF_in = input_grid_TAF->tree().getAccessor();
        auto accessor_TAF_out = output_grid_TAF->tree().getAccessor();
        int incrementos_vecinos[] = {-1,0,1};
        int len_incrementos = 3;
        
        float n_i = 0;
        bool esVecino = false;
        //Se calcula n_i , que determina si se es vecino de una endothelial
        //if(accessor_endothelial.getValue(coord)<threshold_vecino){
            // for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
            //     for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
            //         for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
            //             int incremento_x = incrementos_vecinos[i_incremento_x];
            //             int incremento_y = incrementos_vecinos[i_incremento_y];
            //             int incremento_z = incrementos_vecinos[i_incremento_z];
    
    
            //             if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
            //                 n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            
            //             }
            //             esVecino = n_i > threshold_vecino;//Esto igual esta feo
                        
            //         }
            //     }
            // }
        //}

        float n_c = 0.025;
        //printf("%f\n",n_i);
        esVecino = accessor_endothelial.getValue(coord)==1.0;
        if(esVecino){
            //printf("VECINO\n");
            n_i = 1.0;
        }else{
            n_i  = 0.0;
        }
        float c = leaf_s->getValue(i);
        
        
        float derivative = -n_c * n_i* c;
        // if(c > 0){
        //     printf("%f %f %f\n",c,n_i,derivative);
        // }
        float old_c = leaf_s->getValue(i);
        auto new_value = old_c + derivative * time_factor;
        if(new_value < 0 ){
            new_value = 0 ;
        }
        if(new_value >1 ){
            new_value = 1;
        }
        
        leaf_d->setValueOnly(i,new_value);
        

        

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationFibronectin(nanovdb::FloatGrid* input_grid_endothelial,nanovdb::FloatGrid* input_grid_Fibronectin,nanovdb::FloatGrid* input_grid_MDE,nanovdb::FloatGrid* output_grid_Fibronectin,uint64_t leafCount){
    auto kernel = [input_grid_endothelial,input_grid_Fibronectin,input_grid_MDE,output_grid_Fibronectin] __device__ (const uint64_t n) {
        auto *leaf_d = output_grid_Fibronectin->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = input_grid_Fibronectin->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_mde = input_grid_MDE->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        //Coordenadas del voxel globales
        auto coord = leaf_d->offsetToGlobalCoord(i);
        auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
        auto accessor_Fibronectin_in = input_grid_Fibronectin->tree().getAccessor();
        auto accessor_Fibronectin_out = output_grid_Fibronectin->tree().getAccessor();
        int incrementos_vecinos[] = {-1,0,1};
        int len_incrementos = 3;
        
        float n_i = 0;
        bool esVecino = false;
        //Se calcula n_i , que determina si se es vecino de una endothelial
        //if(accessor_endothelial.getValue(coord)<threshold_vecino){
            // for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
            //     for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
            //         for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
            //             int incremento_x = incrementos_vecinos[i_incremento_x];
            //             int incremento_y = incrementos_vecinos[i_incremento_y];
            //             int incremento_z = incrementos_vecinos[i_incremento_z];
    
    
            //             if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
            //                 n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            
            //             }
            //             esVecino = n_i > threshold_vecino;//Esto igual esta feo
                        
            //         }
            //     }
            // }
        //}
        
        
        float production_rate = 0.0125;
        float degradation_rate = 0.1;
        esVecino = accessor_endothelial.getValue(coord) == 1.0;
        if(esVecino){
            n_i = 1.0;
        }else{
            n_i = 0.0;
        }
        
        float old_f = leaf_s->getValue(i);
        float old_mde = leaf_mde->getValue(i);

        float derivative = production_rate * n_i - degradation_rate * old_f * old_mde;
        // if(derivative > 0 ){
        //     printf("%f\n",old_f + derivative * time_factor);
        // }
        auto new_value = old_f + derivative*time_factor;
        if(new_value < 0 ){
            new_value =  0;
        }
        if(new_value > 1){
            new_value = 1;
        }
        leaf_d->setValueOnly(i,new_value);
        //leaf_d->setValueOnly(i,n_i);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationMDE(nanovdb::FloatGrid* input_grid_endothelial,nanovdb::FloatGrid* input_grid_MDE,nanovdb::FloatGrid* output_grid_MDE,uint64_t leafCount){
    auto kernel = [input_grid_endothelial,input_grid_MDE,output_grid_MDE] __device__ (const uint64_t n) {
        auto *leaf_d = output_grid_MDE->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = input_grid_MDE->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        //Coordenadas del voxel globales
        auto coord = leaf_d->offsetToGlobalCoord(i);
        auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
        auto accessor_Fibronectin_in = input_grid_MDE->tree().getAccessor();
        auto accessor_Fibronectin_out = output_grid_MDE->tree().getAccessor();
        int incrementos_vecinos[] = {-1,0,1};
        int len_incrementos = 3;
        
        float n_i = 0;
        bool esVecino = false;
        // //Se calcula n_i , que determina si se es vecino de una endothelial
        //if(accessor_endothelial.getValue(coord)<threshold_vecino){
            // for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
            //     for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
            //         for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
            //             int incremento_x = incrementos_vecinos[i_incremento_x];
            //             int incremento_y = incrementos_vecinos[i_incremento_y];
            //             int incremento_z = incrementos_vecinos[i_incremento_z];
    
    
            //             if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
            //                 n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                            
            //             }
            //             esVecino = n_i > threshold_vecino;//Esto igual esta feo
                        
            //         }
            //     }
            // }
        //}
        //esVecino = accessor_endothelial.getValue(coord)==1.0;
        float production_rate = 0.0000015;
        
        float diffussion_coefficient = 0.0025;
        float degradation_rate = 0.75;
        //printf("%f %f %f\n",production_rate,diffussion_coefficient,degradation_rate);
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*input_grid_MDE);
        stencilNano.moveTo(coord);
        float laplacian = stencilNano.laplacian();
        esVecino = accessor_endothelial.getValue(coord)==1.0;
        if(esVecino){
            n_i = 1.0;
        }else{
            n_i = 0.0;
        }
        float old_mde = leaf_s->getValue(i);
        float derivative = n_i * production_rate + diffussion_coefficient * laplacian * old_mde - degradation_rate * old_mde;
        // float factor_1 = n_i * production_rate;
        // float factor_2 = diffussion_coefficient * laplacian * old_mde;
        // float factor_3 = degradation_rate * old_mde;
        // if(laplacian!=0){
        //     printf("%f\n",laplacian);
        // }
        
        // if(factor_1 != 0|| factor_2 != 0 || factor_3!=0){
        //     printf("%f %f %f\n",n_i * production_rate,diffussion_coefficient*laplacian*old_mde,degradation_rate*old_mde);

        // }
        // if(derivative > 0 && factor_1 > 0.000002){
        //     printf("%f %f %f\n",factor_1,factor_2,factor_3);
        // }
        //float derivative = diffussion_coefficient * laplacian;
        auto new_value = old_mde + derivative * time_factor;
        if(new_value < 0 ){
            new_value = 0;
        }
        if(new_value >1 ){
            new_value = 1;
        }
        leaf_d->setValueOnly(i,new_value);
        //leaf_d->setValueOnly(i,n_i);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void pruebaGradiente(nanovdb::Vec3fGrid  *grid_d,nanovdb::FloatGrid* gridSource ,uint64_t leafCount)
{
    auto kernel = [grid_d,gridSource] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = gridSource->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf_d->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridSource);
        
        auto accessor = gridSource->tree().getAccessor();
        
        stencilNano.moveTo(coord_nano);
        auto gradiente = stencilNano.gradient();
        // s
        // auto suma = gradiente[0]+gradiente[1]+gradiente[2];
        // if(suma!=0){
        //     nanovdb::Coord coord_aux = coord;
        //     coord_aux[0] = 0 ;
        //     coord_aux[1] = 0 ;
        //     coord_aux[2] = 0 ;
        //     //printf("%f %f\n",suma,leaf_s->getValue(coord));
        //     printf("%f \n",accessor.getValue(coord));
        // }
        leaf_d->setValueOnly(coord,gradiente);
        
        
    };

    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}


__device__ float chemotacticSensivity(float c){
    float chemotacticMigration = 0.38;
    float chemotacticConstant = 0.6;
    return chemotacticMigration /(1 + chemotacticConstant*c);
}
__device__ int getPosition(nanovdb::Coord coord_self,nanovdb::FloatGrid * endothelialTip){
    auto accessor = endothelialTip->tree().getAccessor();
    nanovdb::Coord coord_i = coord_self;

    coord_i[2] = coord_i[2] - 1; //Comprobamos frente
    if(accessor.getValue(coord_i) != 0 ){
        return 6;//coord_self esta a la dch de la celula endothelial
    }
    coord_i[2] = coord_self[2];

    coord_i[2] = coord_i[2] + 1; //Comprobamos frente
    if(accessor.getValue(coord_i) != 0 ){
        return 5;//coord_self esta a la dch de la celula endothelial
    }
    coord_i[2] = coord_self[2];

    coord_i[1] = coord_i[1] +1; //Comprobamos arriba;
    if(accessor.getValue(coord_i) != 0 ){
        return 3;//coord_self esta debajo de la celula endothelial
    }
    coord_i[1] = coord_self[1];

    coord_i[0] = coord_i[0] +1; //Comprobamos derecha
    if(accessor.getValue(coord_i) != 0 ){
        return 4;//coord_self esta a la izq de la celula endothelial
    }
    coord_i[0] = coord_self[0];

    coord_i[1] = coord_i[1] - 1; //Comprobamos abajo
    if(accessor.getValue(coord_i) != 0 ){
        return 1;//coord_self esta arriba de la celula endotelial
    }
    coord_i[1] = coord_self[1];

    coord_i[0] = coord_i[0] - 1; //Comprobamos izquierda
    if(accessor.getValue(coord_i) != 0 ){
        return 2;//coord_self esta a la dch de la celula endothelial
    }
    coord_i[0] = coord_self[0];

    
    return 0;


}
__device__ bool  isNextToEndothelial(nanovdb::Coord coord,nanovdb::FloatGrid * input_grid_endothelial){
    auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
    int incrementos_vecinos[] = {-1,0,1};
    int len_incrementos = 3;
    float n_i = 0;
    bool esVecino = false;
    //Se calcula n_i , que determina si se es vecino de una endothelial
    //if(accessor_endothelial.getValue(coord)<threshold_vecino){
    //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
    for(int i_incremento_x = 0;i_incremento_x<len_incrementos && !esVecino;i_incremento_x++){
        for(int i_incremento_y = 0 ;i_incremento_y<len_incrementos && !esVecino;i_incremento_y++){
            for(int i_incremento_z = 0 ;i_incremento_z<len_incrementos && !esVecino;i_incremento_z++){
                int incremento_x = incrementos_vecinos[i_incremento_x];
                int incremento_y = incrementos_vecinos[i_incremento_y];
                int incremento_z = incrementos_vecinos[i_incremento_z];


                if(accessor_endothelial.isActive(coord.offsetBy(incremento_x,incremento_y,incremento_z))){
                    n_i = accessor_endothelial.getValue(coord.offsetBy(incremento_x,incremento_y,incremento_z));
                    esVecino = n_i > threshold_vecino;//Esto igual esta feo
                }else{
                    esVecino = false;
                }
                
                
            }
        }
    }
    return esVecino;
}
__device__ float average(nanovdb::Coord coord,nanovdb::FloatGrid * input_grid,uint64_t n){
    auto accessor_endothelial = input_grid->tree().getAccessor();
    auto* leaf = input_grid->tree().getFirstNode<0>() + (n >> 9);
    //int desplazamientos[] = {-4,-3,-2,-1,0,1,2,3,4};
    int desplazamientos[] = {-1,0,1};
    int len_desp = 3;
    float n_i = 0;
    bool esVecino = false;
    float total = 0.0;
    float accum = 0.0;
    //Se calcula n_i , que determina si se es vecino de una endothelial
    //if(accessor_endothelial.getValue(coord)<threshold_vecino){
    //printf("%d %d %d\n",coord[0],coord[1],coord[2]);
    

    
    for(int dimension = 0 ;dimension <3;dimension++){

            for(int desplazamiento = 0;desplazamiento<len_desp;desplazamiento++){
                nanovdb::Coord new_coord = coord;
                new_coord[dimension] += desplazamientos[desplazamiento];
                if(accessor_endothelial.isActive(new_coord)){
                    float value_i = accessor_endothelial.getValue(new_coord);
                    total = total + 1.0;
                    accum = accum + value_i;
                }
                
                //printf("Position Self %d, new_coord %d %d %d value_i %f\n",positionSelf,new_coord[0],new_coord[1],new_coord[2],value_i);
                //printf("Value i %f\n",value_i);
                
            }
    
        }
    
    if(total == 0.0){
        total = 1.0;
    }
    // if(accum > 0.0 ){
    //     printf("accum: %f\n",accum);
    // }
    //return 1;
    return accum / total;
}
__device__ bool isNextToEndothelialDiscrete(nanovdb::Coord coord,nanovdb::FloatGrid * input_grid_endothelial){
    auto accessor_endothelial = input_grid_endothelial->tree().getAccessor();
    int desplazamientos[] = {-1,1};
    int len_desp = 2;
    
    //int desplazamiento_max[3];
    
    
    //nanovdb::Coord coord_max = coord;
    //float max_derivative = computeEndothelial(coord,stencilEndothelial,stencilTAF,stencilFibronectin);

    //Se calcula el maximo
    bool esVecino = false;
    for(int dimension = 0 ;dimension <3&&!esVecino;dimension++){

        for(int desplazamiento = 0;desplazamiento<len_desp&&!esVecino;desplazamiento++){
            nanovdb::Coord new_coord = coord;
            new_coord[dimension] += desplazamientos[desplazamiento];
            float value_i = accessor_endothelial.getValue(new_coord);
            esVecino = value_i > threshold_vecino;
        }
    }
    return esVecino;
}
/*
    Ecuacion 6
*/
void equationEndothelial(nanovdb::FloatGrid * grid_s,nanovdb::FloatGrid * grid_d,nanovdb::FloatGrid* gridTAF,nanovdb::FloatGrid* gridFibronectin,nanovdb::Vec3fGrid* gradientTAF,nanovdb::Vec3fGrid* gradientFibronectin,nanovdb::FloatGrid* gridTip,uint64_t leafCount){
    auto kernel = [grid_s,grid_d,gridTAF,gridFibronectin,gradientTAF,gradientFibronectin,gridTip] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = grid_s->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_taf = gridTAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf_d->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        /*
            Primera parte: Difusion aleatoria
        */
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*grid_s);
        stencilNano.moveTo(coord_nano);
        float old_n = leaf_s->getValue(coord_nano);
        float laplacian = stencilNano.laplacian();
        //printf("%f\n",laplacian);
        float factorEndothelial = laplacian * 0.0003 ;
        /*
            Segunda parte, chimiotaxis TAF
        */
        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencilTAF(*gradientTAF);
        
        stencilTAF.moveTo(coord_nano);
        auto gradientTAF = stencilTAF.gradient();
        float taf_value = leaf_taf->getValue(coord_nano);
        for(int index = 0 ;index <3;index++){
             gradientTAF[index] *= chemotacticSensivity(taf_value);
             gradientTAF[index] *= old_n;
        }
        float factorTAF = gradientTAF[0][0] + gradientTAF[1][1] + gradientTAF[2][2];
        
        //factorTAF *=10;
        //printf("%f  %f  %f\n",gradientTAF[0][0],gradientTAF[1][1],gradientTAF[2][2]);
        
        //float factorTAF  = stencilNano.gaussianCurvature() ;
        /*
            Tercera parte, Fibronectin
        */
        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencilFibronectin(*gradientFibronectin);
        stencilFibronectin.moveTo(coord_nano);
        auto gradientFibronectin = stencilFibronectin.gradient();
        float factorFibronectin = gradientFibronectin[0][0] + gradientFibronectin[1][1] + gradientFibronectin[2][2];
        factorFibronectin = factorFibronectin * 0.28;
        
        //printf("%f %f %f\n",factorEndothelial,factorTAF,factorFibronectin);

        //float derivative = factorEndothelial  + factorTAF ;//+ factorFibronectin;
        float derivative = factorEndothelial - factorTAF - factorFibronectin;
        
        // if(derivative > 0 ){
        //     printf("%f\n",derivative);
        // }
        // if(derivative > 100){
        //     printf("%f %f %f\n",factorEndothelial,factorTAF,factorFibronectin);
        // }
        //float derivative = -factorTAF;
        int positionSelf = getPosition(coord_nano,gridTip);
        auto new_value = old_n + derivative * time_factor;
        if(isNextToEndothelialDiscrete(coord_nano,gridTip)){
            if(derivative != 0){
                //printf("endo:%f taf:%f fibro:%f new:%f positionSelf:%d coord: %d %d %d\n",factorEndothelial,factorTAF,factorFibronectin,new_value,positionSelf,coord_nano[0],coord_nano[1],coord_nano[2]);
            }
        }
        
        if(new_value < 0 ){
            new_value = 0 ;
        }
        if(new_value > 1){
            new_value = 1;
        }
        
        leaf_d->setValueOnly(coord_nano,new_value);//6 minutos //* 60 segundos
        //leaf_d->setValueOnly(coord_nano,derivative);//6 minutos //* 60 segundos

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
__device__ float computeEndothelial(nanovdb::Coord coord_nano,nanovdb::CurvatureStencil<nanovdb::FloatGrid>& stencilEndothelial,nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> &stencilTAF,nanovdb::CurvatureStencil<nanovdb::Vec3fGrid>& stencilFibronectin){
    /*
    Primera parte: Difusion aleatoria
    */
    
    stencilEndothelial.moveTo(coord_nano);
    //float old_n = leaf_s->getValue(coord_nano);
    float laplacian = stencilEndothelial.laplacian();
    //printf("%f\n",laplacian);
    float factorEndothelial = laplacian * 0.0003 ;

    /*
    Segunda parte, chimiotaxis TAF
    */
    
    
    stencilTAF.moveTo(coord_nano);
    auto gradientTAF = stencilTAF.gradient();
    float factorTAF = gradientTAF[0][0] + gradientTAF[1][1] + gradientTAF[2][2];
    /*
    Tercera parte, Fibronectin
    */
    
    stencilFibronectin.moveTo(coord_nano);
    auto gradientFibronectin = stencilFibronectin.gradient();
    float factorFibronectin = gradientFibronectin[0][0] + gradientFibronectin[1][1] + gradientFibronectin[2][2];
    factorFibronectin = factorFibronectin * 0.28;
    
    //printf("%f %f\n",factorTAF,factorFibronectin);

    float derivative = factorEndothelial  - factorTAF - factorFibronectin;
    //derivative = -factorTAF-factorFibronectin;
    derivative = -factorTAF;
    return derivative;
}


__device__ bool isMax(nanovdb::Coord coord_self,nanovdb::FloatGrid * endothelialTip,nanovdb::FloatGrid * endothelial,nanovdb::FloatGrid* endothelialDiscrete){
    auto accessor_endothelial = endothelial->tree().getAccessor();
    auto accessor_discrete = endothelialDiscrete->tree().getAccessor();
    int positionSelf = getPosition(coord_self,endothelialTip);
    //printf("%d\n",positionSelf);
    nanovdb::Coord coord_endothelial = coord_self;
    switch(positionSelf){
        case 1:
            coord_endothelial[1] -=1;
            break;
        case 2:
            coord_endothelial[0] -= 1;
            break;
        case 3:
            coord_endothelial[1] += 1;
            break;
        case 4:
            coord_endothelial[0] += 1;
            break;
        case 5:
            coord_endothelial[2] += 1;
            break;
        case 6:
            coord_endothelial[2] -= 1;
            break;
        default:
            break;

    };
    int desplazamientos[] = {-1,1};
    int len_desp = 2;
    
    nanovdb::Coord coord_max;
    float value_max = -1;
    if(positionSelf != 0 ){
        for(int dimension = 0 ;dimension <3;dimension++){

            for(int desplazamiento = 0;desplazamiento<len_desp;desplazamiento++){
                nanovdb::Coord new_coord = coord_endothelial;
                new_coord[dimension] += desplazamientos[desplazamiento];
                float value_i = accessor_endothelial.getValue(new_coord);
                //printf("Position Self %d, new_coord %d %d %d value_i %f\n",positionSelf,new_coord[0],new_coord[1],new_coord[2],value_i);
                //printf("Value i %f\n",value_i);
                if(accessor_discrete.getValue(new_coord) == 0.0 && value_i > value_max){
                    value_max = value_i;
                    coord_max = new_coord;
                }
                if(positionSelf == 1){
                    //printf("desplazamiento %d dimension %d discrete %f value_i %f value_max %f new coord %d %d %d\n",
                    //desplazamiento,dimension,accessor_discrete.getValue(new_coord),value_i,value_max,new_coord[0],new_coord[1],new_coord[2]);
                }
            }
     
        }
    }
    float endothelialSelf = accessor_endothelial.getValue(coord_self);
    //printf("PositionSelf %d, coord_self %d %d %d Coord max %d %d %d value Self %f\n",positionSelf,coord_self[0],coord_self[1],coord_self[2],coord_max[0],coord_max[1],coord_max[2],endothelialSelf);
    //printf("Value_max %f\n",value_max);
    //return positionSelf == 4;
    return positionSelf != 0 && coord_max == coord_self;


}

__device__ void moveRandom(nanovdb::Coord coord_self,nanovdb::FloatGrid* gridEndothelial,nanovdb::FloatGrid* gridEndothelialDiscrete,nanovdb::FloatGrid* gridTip,float randomValue,int n ){
    auto accessor_endothelial = gridEndothelial->tree().getAccessor();
    auto accessor_discrete = gridEndothelialDiscrete->tree().getAccessor();
    auto accessor_tip = gridTip->tree().getAccessor();
    auto *leaf_tip = gridTip->tree().getFirstNode<0>()+(n>>9);
    auto *leaf_discrete = gridEndothelialDiscrete->tree().getFirstNode<0>()+(n>>9);
    int positionSelf = getPosition(coord_self,gridTip);
    //printf("%d\n",positionSelf);
    nanovdb::Coord coord_endothelial = coord_self;
    switch(positionSelf){
        case 1:
            coord_endothelial[1] -=1;
            break;
        case 2:
            coord_endothelial[0] -= 1;
            break;
        case 3:
            coord_endothelial[1] += 1;
            break;
        case 4:
            coord_endothelial[0] += 1;
            break;
        case 5:
            coord_endothelial[2] += 1;
            break;
        case 6:
            coord_endothelial[2] -= 1;
            break;
        default:
            break;

    };
    if(positionSelf == 0 ){
        return;
    }


    int desplazamientos[] = {-1,1};
    const int len_desp = 2;
    
    nanovdb::Coord coord_max;
    float value_accum = 0.0;
    const int length = 3 * len_desp;
    float values[length];
    nanovdb::Coord coords[length];
    int i_value = 0;
    if(positionSelf != 0 ){
        for(int dimension = 0 ;dimension <3;dimension++){

            for(int desplazamiento = 0;desplazamiento<len_desp;desplazamiento++){
                nanovdb::Coord new_coord = coord_endothelial;
                new_coord[dimension] += desplazamientos[desplazamiento];
                float value_i = accessor_endothelial.getValue(new_coord);
                //printf("Position Self %d, new_coord %d %d %d value_i %f\n",positionSelf,new_coord[0],new_coord[1],new_coord[2],value_i);
                value_accum += value_i;
                values[i_value] = value_i;
                coords[i_value] = new_coord;
                i_value++;
            }
        }
    }
    // for(int i = 0 ;i<length;i++){
    //     values[i] = 1-values[i] / value_accum;
    // } 
    thrust::sort(thrust::device, values, values + length);
    bool decided = false;
    for(int i = 0 ;i<length && !decided;i++){
        if(coords[i]==coord_self && randomValue>values[i]){
            leaf_tip->setValue(coords[i],2);
            leaf_discrete->setValue(coords[i],1);
            decided = true;
        }
    }


}

void equationEndothelialDiscrete(nanovdb::FloatGrid * grid_source_discrete,nanovdb::FloatGrid * grid_destiny_discrete,nanovdb::FloatGrid* gridDerivativeEndothelial,nanovdb::FloatGrid* gridDerivativeEndothelialWrite,nanovdb::FloatGrid* gridTAF,nanovdb::FloatGrid * gridTipRead,nanovdb::FloatGrid* gridTipWrite,int seed,uint64_t leafCount){
    
    auto kernel = [grid_source_discrete,grid_destiny_discrete,gridDerivativeEndothelial,gridDerivativeEndothelialWrite,gridTAF,gridTipRead,gridTipWrite,seed] __device__ (const uint64_t n) {
        auto *leaf_d = grid_destiny_discrete->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_s = grid_source_discrete->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_tip_write = gridTipWrite->tree().getFirstNode<0>()+(n>>9);
        auto *leaf_tip_read = gridTipRead->tree().getFirstNode<0>()+(n>>9);
        auto *leaf_endothelial_write = gridDerivativeEndothelialWrite->tree().getFirstNode<0>()+(n>>9);
        auto *leaf_endothelial_read = gridDerivativeEndothelial->tree().getFirstNode<0>()+(n>>9);
        //auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        

        auto coord = leaf_tip_write->offsetToGlobalCoord(i);
        auto coord_d = leaf_d->offsetToGlobalCoord(i);
        //float taf_value = leaf_TAF->getValue(coord);

        // if(coord[0] == 0&&coord[1]==0 && coord[2]==0){
        //     printf("RAndom %f\n",random);

        // }        

        //leaf_d->setValue(coord,random);

        // nanovdb::Coord coord_dummy;
        // coord_dummy[0] = 0 ;
        // coord_dummy[1] =0 ;
        // coord_dummy[2] = 0 ;
        //leaf_d->setValueOnly(coord_dummy,100);
        // if(leaf_tip_read->getValue(i)>0){
        //     printf("TIP\n");
        // }
       // static int first = true;
       if(isNextToEndothelialDiscrete(coord_d,gridTipRead)){
        //if(leaf_tip->getValue(i)>0){
            //if((coord_d[1]-1)%2 == 0){
            //if(coord_d[1]%2 == 0 ){
            int positionSelf = getPosition(coord_d,gridTipRead);
            //moveRandom(coord,gridDerivativeEndothelial,grid_destiny_discrete,gridTipWrite,random,n);
            //leaf_d->setValue(coord_d,1.0);
            auto coord_left = coord_d;
            coord_left[0]-=1;
            float tip_left = leaf_tip_read->getValue(coord_left);
            //printf("Next, endothelial: %f ,position:%d , tip left:%f\n",leaf_endothelial_read->getValue(coord_d),positionSelf,tip_left);

            if(isMax(coord_d,gridTipRead,gridDerivativeEndothelial,grid_source_discrete)){
                if(leaf_s->getValue(coord)>0.0){
                    printf("WTF\n");
                }

                printf("Is max %d \n",positionSelf);
                leaf_d->setValue(coord_d,1.0);
                leaf_tip_write->setValue(coord_d,2.0);
                //leaf_endothelial_write->setValue(coord_d,1.0);
                
            
            }
            
            // leaf_d->setValue(coord_d,1.0);
            // //leaf_s->setValue(coord_d,1.0);
            // if(leaf_s->getValue(coord_d)==0){
            //     leaf_tip_write->setValue(coord_d,2.0);
            // }
            //leaf_tip->setValue(coord_d,0);
            //coord_d[0]+=1;
            //leaf_tip->setValue(coord_d,1);

            //FALTA EL BRANCHING
           
        }else{
            float value = leaf_s->getValue(i);
            leaf_d->setValue(coord_d,value);
        }
        // }else if(false&&isNextToEndothelialDiscrete(coord_d,grid_source_discrete)){
        //     //first = false;
        //     if(taf_value >= 0.8 && random >= 1.0-vector_probabilidades[3]){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else if(taf_value >=0.7 && random >= 1-vector_probabilidades[2] ){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else if(taf_value >= 0.5&& random >= 1-vector_probabilidades[1]){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else if(taf_value >=0.3&& random >= 1-vector_probabilidades[0]){
        //         //printf("NEW TIP\n");
        //         leaf_tip_write->setValueOnly(coord,1.0);
        //         leaf_d->setValueOnly(coord,1.0);
        //     }else{
        //         //NO hay branch
        //         //leaf_tip->setValueOnly(coord,0.0);
        //     }
        //     //leaf_d->setValueOnly(coord,1.0);
        // }else{
        //     leaf_tip_write->setValue(coord_d,0);
        // }
        
       
        //leaf_d->setValueOnly(i,0.0);
        
        
    
    
    
    
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void branching(nanovdb::FloatGrid* gridEndothelialTip,nanovdb::FloatGrid* gridTAF,int seed,int leafCount){
    auto kernel = [gridEndothelialTip,gridTAF,seed] __device__ (const uint64_t n) {
        auto* leaf_tip = gridEndothelialTip->tree().getFirstNode<0>() + (n >> 9);
        auto* leaf_taf = gridTAF->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;

        thrust::minstd_rand rng;
        thrust::default_random_engine randEng;
        thrust::uniform_real_distribution<float> uniDist;
        int discard = seed+n;
        randEng.discard(discard);
        float random = uniDist(randEng);

        auto coord_d = leaf_tip->offsetToGlobalCoord(i);
        float value = leaf_tip->getValue(i);
        float taf_value = leaf_taf->getValue(i);
        float new_value = 1;
        float vector_probabilidades[] = {0.04,0.06,0.08,0.2};
        if(value == 1.0){
            if(taf_value >= 0.8 && random >= 1.0-vector_probabilidades[3]){
                printf("NEW TIP\n");
                // leaf_tip_write->setValueOnly(coord,1.0);
                // leaf_d->setValueOnly(coord,1.0);
            }else if(taf_value >=0.7 && random >= 1-vector_probabilidades[2] ){
                printf("NEW TIP\n");
                // leaf_tip_write->setValueOnly(coord,1.0);
                // leaf_d->setValueOnly(coord,1.0);
            }else if(taf_value >= 0.5&& random >= 1-vector_probabilidades[1]){
                printf("NEW TIP\n");
                // leaf_tip_write->setValueOnly(coord,1.0);
                // leaf_d->setValueOnly(coord,1.0);
            }else if(taf_value >=0.3&& random >= 1-vector_probabilidades[0]){
                printf("NEW TIP\n");
                // leaf_tip_write->setValueOnly(coord,1.0);
                // leaf_d->setValueOnly(coord,1.0);
            }else{
                //NO hay branch
                leaf_tip->setValue(coord_d,0.0);
                new_value = 0;
                //leaf_tip->setValueOnly(coord,0.0);
            }
        }else{
            new_value = value -1;
        }
        

        // if(true || i%2==0){
        //     new_value = value-1;
        // }
        //float new_value = value-1;
        if(new_value < 0 ){
            new_value =0;
        }
        if(new_value != 0.0 && new_value != 1.0){
            printf("%f\n",new_value);
        }
        //new_value = random;
        leaf_tip->setValue(coord_d,new_value);

        
        

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}


/*
    Genera el gradiente escalado del TAF, para poder calcular la divergencia
*/
void generateGradientTAF(nanovdb::FloatGrid * gridTAF,nanovdb::FloatGrid * gridTAFEndothelial,nanovdb::Vec3fGrid* gradientTAF,uint64_t leafCount){
    auto kernel = [gridTAF,gridTAFEndothelial,gradientTAF] __device__ (const uint64_t n) {
        auto *leaf_s = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAFEndothelial = gridTAFEndothelial->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Gradient = gradientTAF->tree().getFirstNode<0>() + (n >> 9);
        auto accessor_aux = gradientTAF->getAccessor();
        const int i = n & 511;
        auto coord = leaf_s->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridTAFEndothelial);
        //printf("%d %d %d\n",coord_nano[0],coord_nano[1],coord_nano[2]);
        stencilNano.moveTo(coord_nano);
        auto gradient = stencilNano.gradient();
        float sensivity = chemotacticSensivity(leaf_s->getValue(i));
        
        gradient = gradient *sensivity;
        // if(coord[0]== 0 || coord[1]==0||coord[2]==0){
        //     gradient[0] = 0;
        //     gradient[1] = 0 ;
        //     gradient[2] = 0;
        // }
        // if(gradient[0]!=0 || gradient[1]!=0|| gradient[2]!=0){
        //     printf("<%f %f %f>  sensivity:%f\n",gradient[0],gradient[1],gradient[2],sensivity);
        // }
        
        
        //gradient[0] = 2.0;
        //gradient[1] = 1.0 ;
        //gradient[2] = 3.0;
        leaf_Gradient->setValueOnly(i,gradient);
        //auto aux = accessor_aux.getValue(coord);
        //printf("%f %f %f\n",aux[0],aux[1],aux[2]);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

/*
    Genera el gradiente escalado de la Fibronectina, para poder calcular la divergencia
*/
void generateGradientFibronectin(nanovdb::FloatGrid * gridFibronectin,nanovdb::FloatGrid * gridEndothelial,nanovdb::Vec3fGrid* gradientFibronectin,uint64_t leafCount){
    auto kernel = [gridFibronectin,gridEndothelial,gradientFibronectin] __device__ (const uint64_t n) {
        auto *leaf_s = gridFibronectin->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Gradient = gradientFibronectin->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        auto coord = leaf_s->offsetToGlobalCoord(i);
        const nanovdb::Coord coord_nano = coord;
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridFibronectin);
        stencilNano.moveTo(coord_nano);
        auto gradient = stencilNano.gradient();
        
        float endothelialValue = leaf_Endothelial->getValue(i);
        gradient = gradient  * endothelialValue;
        
        leaf_Gradient->setValueOnly(coord,gradient);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void divergence(nanovdb::Vec3fGrid *grid_s,nanovdb::FloatGrid *grid_d,uint64_t leafCount){
    auto kernel = [grid_s,grid_d] __device__ (const uint64_t n) {
        auto *leaf_s = grid_s->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        auto coord = leaf_s->offsetToGlobalCoord(i);

        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencil(*grid_s);
        stencil.moveTo(coord);
        auto gradient = stencil.gradient();
        auto divergence = gradient[0][0]+gradient[1][1] + gradient[2][2];
        // if(gradient[0][0]!= 0 && divergence != 0 ){
        //     printf("%f\n",divergence);
        // }
        leaf_d->setValueOnly(coord,divergence);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
void laplacian(nanovdb::FloatGrid * grid_s,nanovdb::FloatGrid * grid_d, uint64_t leafCount){
    auto kernel = [grid_s,grid_d] __device__ (const uint64_t n) {
        auto *leaf_s = grid_s->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        auto coord = leaf_s->offsetToGlobalCoord(i);

        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencil(*grid_s);
        const nanovdb::Coord coord_nano = coord;
        stencil.moveTo(coord_nano);
        auto old_value = leaf_s->getValue(i);
        auto laplacian = stencil.laplacian();
        if(laplacian < 0.0){
            laplacian=0.0;
        }
        // if(laplacian!= 0){
        //     printf("%f\n",laplacian);
        // }
        auto new_value = old_value + laplacian*0.2;

        // if(new_value < 0 ){
        //     new_value = 0;
        // }
        
        leaf_d->setValueOnly(i,new_value);
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void product(nanovdb::FloatGrid * gridTAF,nanovdb::FloatGrid * gridEndothelial,nanovdb::FloatGrid *grid_d, uint64_t leafCount){
    auto kernel = [gridTAF,gridEndothelial,grid_d] __device__ (const uint64_t n) {
        auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        //auto coord = leaf_d->offsetToGlobalCoord(i);
        auto endothelial = leaf_Endothelial->getValue(i);
        auto taf = leaf_TAF->getValue(i);
        
        auto new_value = leaf_TAF->getValue(i)*leaf_Endothelial->getValue(i);
        // if(endothelial != 0){
        //     printf("end:%f taf:%f newVal:%f\n",endothelial,taf,new_value);
        // }
        new_value = leaf_TAF->getValue(i);
        //new_value = leaf_Endothelial->getValue(i);
        leaf_d->setValueOnly(i,new_value);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void cleanEndothelial(nanovdb::FloatGrid * gridEndothelial,uint64_t leafCount){
    auto kernel = [gridEndothelial] __device__ (const uint64_t n) {
        auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;

        auto coord = leaf_Endothelial->offsetToGlobalCoord(i);
        // if(coord[1]<-149){
        //     printf("%d\n",coord[1]);
        // }
        if(coord[0] == -250 || coord[1] == -150 || coord[2] == -250){
            //leaf_Endothelial->setValueOnly(i,0.00001);
            leaf_Endothelial->setValueOnly(i,0.1);
            //printf("Uese\n");
        }
        if(coord[0] == 0 || coord[1] == 0 || coord[2] == 0){
            //leaf_Endothelial->setValueOnly(i,0.00001);
            leaf_Endothelial->setValueOnly(i,0.1);
            //printf("Uese\n");
        }
        // if(coord[0] <= -230 || coord[1] <= -130 || coord[2] <= -230){
        //     //leaf_Endothelial->setValueOnly(i,0.00001);
        //     leaf_Endothelial->setValueOnly(i,0);
        //     //printf("Uese\n");
        // }
        // if(coord[0] >= -20 || coord[1] >= -20 || coord[2] >= -20){
        //     //leaf_Endothelial->setValueOnly(i,0.00001);
        //     leaf_Endothelial->setValueOnly(i,0);
        //     //printf("Uese\n");
        // }
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void normalize(nanovdb::FloatGrid * gridTAF,float maxValue,float prevMax, uint64_t leafCount){
    auto kernel = [gridTAF,maxValue,prevMax] __device__ (const uint64_t n) {
       // auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        //auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        //auto coord = leaf_d->offsetToGlobalCoord(i);

        
        float new_value = leaf_TAF->getValue(i);
        if(maxValue > 0 ){
            new_value = (1.0 - new_value / maxValue)*prevMax;
        }
        leaf_TAF->setValueOnly(i,new_value);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
void addMax(nanovdb::FloatGrid * gridTAF, float maxValue,uint64_t leafCount){
    auto kernel = [gridTAF,maxValue] __device__ (const uint64_t n) {
       // auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        //auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        //auto coord = leaf_d->offsetToGlobalCoord(i);

        
        float new_value = leaf_TAF->getValue(i) + maxValue;
        
        leaf_TAF->setValueOnly(i,new_value);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void absolute(nanovdb::FloatGrid * gridTAF, uint64_t leafCount){
    auto kernel = [gridTAF] __device__ (const uint64_t n) {
       // auto *leaf_d = grid_d->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        auto *leaf_TAF = gridTAF->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true
        //auto *leaf_Endothelial = gridEndothelial->tree().getFirstNode<0>() + (n >> 9);// this only works if grid->isSequential<0>() == true

        const int i = n & 511;
        
        //auto coord = leaf_d->offsetToGlobalCoord(i);

        
        float new_value = leaf_TAF->getValue(i) ;
        new_value = -1.0 * new_value;
        
        leaf_TAF->setValueOnly(i,new_value);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void regenerateEndothelial(nanovdb::FloatGrid* gridEndothelialContinue,nanovdb::FloatGrid* gridEndothelialDiscrete,u_int64_t leafCount){
    auto kernel = [gridEndothelialContinue,gridEndothelialDiscrete] __device__ (const uint64_t n) {
        auto *leaf_Endo = gridEndothelialContinue->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Discrete = gridEndothelialDiscrete->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf_Endo->offsetToGlobalCoord(i);
        if(leaf_Discrete->getValue(i)>0.0){
            leaf_Endo->setValue(coord,1.0);
        }
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationBplusSimple(nanovdb::FloatGrid* gridTumor,nanovdb::FloatGrid* gridBplus,nanovdb::FloatGrid* gridOxygen,u_int64_t leafCount){
    auto kernel = [gridTumor,gridBplus,gridOxygen] __device__ (const uint64_t n) {
        auto *leaf_Tumor = gridTumor->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Bplus = gridBplus->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Oxygen = gridOxygen->tree().getFirstNode<0>() + (n >> 9);
        
        const int i = n & 511;
        
        auto coord = leaf_Tumor->offsetToGlobalCoord(i);
        float oxygen = leaf_Oxygen->getValue(i);
        float oxygenThreshold = 0.1;
        
        
        if(oxygen>oxygenThreshold){
            float c_max = 2.0;
            float TtcProliferation= 10.0 * 5.0;
            float tumor_cells = leaf_Tumor->getValue(i);
            float new_value = 0 ;
            new_value = 1.0/TtcProliferation * tumor_cells * (1.0-tumor_cells/c_max);
            // if(new_value > 0.0){
            //     printf("%f\n",new_value);
            // }
            leaf_Bplus->setValue(coord,new_value);
            //leaf_Bplus->setValue(coord,1.0*tumor_cells);
        }else{
            leaf_Bplus->setValue(coord,0.0);
        }

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
void equationBminusSimple(nanovdb::FloatGrid* gridTumor,nanovdb::FloatGrid* gridBminus,nanovdb::FloatGrid* gridOxygen,u_int64_t leafCount){
    auto kernel = [gridTumor,gridBminus,gridOxygen] __device__ (const uint64_t n) {
        auto *leaf_Tumor = gridTumor->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Bminus = gridBminus->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Oxygen = gridOxygen->tree().getFirstNode<0>() + (n >> 9);
        
        const int i = n & 511;
        
        auto coord = leaf_Tumor->offsetToGlobalCoord(i);
        float oxygen = leaf_Oxygen->getValue(i);
        float oxygenThreshold = 0.01;
        
        
        if(oxygen<oxygenThreshold){
            float TtcDeath = 100;
            float tumor_cells = leaf_Tumor->getValue(i);
            float new_value = 0.0;

            new_value = -1.0/TtcDeath * tumor_cells;
            leaf_Bminus->setValue(coord,new_value);
        }else{
            leaf_Bminus->setValue(coord,0.0);
        }

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationPressure(nanovdb::FloatGrid* gridTumor,nanovdb::FloatGrid* gridPressure,u_int64_t leafCount){
    auto kernel = [gridTumor,gridPressure] __device__ (const uint64_t n) {
        auto *leaf_Tumor = gridTumor->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Pressure = gridPressure->tree().getFirstNode<0>() + (n >> 9);

        const int i = n & 511;
        
        auto coord = leaf_Tumor->offsetToGlobalCoord(i);
       

        float cbNorm = 1.0;
        float cbMax = 2.0;

        float tumor_cells = leaf_Tumor->getValue(i);

        if(tumor_cells >= cbMax){
            //Do nothing ????
            //int a = 0 ;
            float value = leaf_Pressure->getValue(coord);
            float new_value = 0.0;

            new_value = (tumor_cells - cbNorm)/(cbMax-cbNorm);
            // if(value != 0 ){
            //     printf("%f\n",value);
            // }
            leaf_Pressure->setValue(coord,1.0);
        }else if(tumor_cells>=cbNorm){
            float new_value = 0.0;

            new_value = (tumor_cells - cbNorm)/(cbMax-cbNorm);
            // if(new_value < 0 ){
            //     new_value = 0.0;
            // }
            leaf_Pressure->setValue(coord,new_value);
        }else{
            leaf_Pressure->setValue(coord,0.0);
        }

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationTumorSimple(nanovdb::Vec3fGrid* gridFlux,nanovdb::FloatGrid* gridBplus,nanovdb::FloatGrid* gridBminus,nanovdb::FloatGrid* gridTumorRead,nanovdb::FloatGrid* gridTumorWrite,u_int64_t leafCount){
    auto kernel = [gridFlux,gridBplus,gridBminus,gridTumorRead,gridTumorWrite] __device__ (const uint64_t n) {
        auto *leaf_Bplus = gridBplus->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Bminus = gridBminus->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Flux = gridFlux->tree().getFirstNode<0>() + (n >> 9);
        auto leaf_tumor_read = gridTumorRead->tree().getFirstNode<0>() + (n>>9);
        auto leaf_tumor_write = gridTumorWrite->tree().getFirstNode<0>() + (n>>9);

        const int i = n & 511;
        
        auto coord = leaf_Flux->offsetToGlobalCoord(i);
       
        
        nanovdb::CurvatureStencil<nanovdb::Vec3fGrid> stencilNano(*gridFlux);
        
        
        
        stencilNano.moveTo(coord);
        auto gradient = stencilNano.gradient();
        auto divergence = gradient[0][0]+gradient[1][1] + gradient[2][2];
        auto value_flux = leaf_Flux->getValue(coord);
        // if(value_flux[0]!=0.0||value_flux[1]!=0.0||value_flux[2]){
        //     printf("gradient matrix %f %f %f ; %f %f %f ; %f %f %f\n",gradient[0][0],gradient[0][1],gradient[0][2],gradient[1][0],gradient[1][1],
        //                                                             gradient[1][2],gradient[2][0],gradient[2][1],gradient[2][2]);
        // }
        float old_m = leaf_tumor_read->getValue(i);
        float b_plus = leaf_Bplus->getValue(i);
        float b_minus = leaf_Bminus->getValue(i);
        float factor_divergence = -divergence * old_m * 0.000001;
        if(factor_divergence<0){
            factor_divergence = 0 ;
        }
        if(factor_divergence!=0){
            printf("%f %f\n",factor_divergence,b_plus);
        }
        float derivative = factor_divergence + b_plus ;//+ b_minus;
        // if(derivative < 0 ) {
        //     derivative  = 0 ;
        // }
        float new_value = old_m + derivative * time_factor;
        if(new_value > 2.0){
            new_value =2.0 ;
        }
        if(new_value < 0.0){
            new_value  = 0.0;
        }
        // if(b_plus != 0.0){
        //     printf("diver:%f plus:%f minus:%f new:%f\n",divergence,b_plus,b_minus,new_value);

        // }
        // if(new_value != 0.0){
        //     printf("new value tumor:%f\n",new_value);
        // }
        leaf_tumor_write->setValue(coord,new_value);
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void equationFluxSimple(nanovdb::FloatGrid* gridPressure,nanovdb::FloatGrid* gridTumor,nanovdb::Vec3fGrid* gridFlux,u_int64_t leafCount){
    auto kernel = [gridPressure,gridTumor,gridFlux] __device__ (const uint64_t n) {
        auto *leaf_Pressure = gridPressure->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_Tumor = gridTumor->tree().getFirstNode<0>() + (n>>9);
        auto *leaf_Flux = gridFlux->tree().getFirstNode<0>() + (n>>9);
        const int i = n & 511;
        
        auto coord = leaf_Tumor->offsetToGlobalCoord(i);
       
        nanovdb::CurvatureStencil<nanovdb::FloatGrid> stencilNano(*gridPressure);
        stencilNano.moveTo(coord);
        auto gradiente = stencilNano.gradient();
        float diffussion_coefficient =.1;//Esto dependera de cada capa de la piel
        // if(leaf_Pressure->getValue(coord)>0.0){
        //     printf("%f %f %f\n",gradiente[0],gradiente[1],gradiente[2]);
        // }

        float tumor_cells = leaf_Tumor->getValue(i);
        for(int i = 0;i<0;i++){
            gradiente[i] *= -diffussion_coefficient* tumor_cells;
        }
        // if(gradiente[0]!=0 || gradiente[1] != 0 || gradiente[2]!=0){
        //     printf("%f %f %f \n",gradiente[0],gradiente[1],gradiente[2]);
        // }
        leaf_Flux->setValue(coord,gradiente);

    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

void discretize(nanovdb::FloatGrid* grid,u_int64_t leafCount){
    auto kernel = [grid] __device__ (const uint64_t n) {
        auto *leaf = grid->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf->offsetToGlobalCoord(i);
        float value = leaf->getValue(i);
        if(value > 0.0){
            value  = 1.0;
        }
        leaf->setValue(coord,value);
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
void average(nanovdb::FloatGrid* grid,nanovdb::FloatGrid* destiny,u_int64_t leafCount){
    auto kernel = [grid,destiny] __device__ (const uint64_t n) {
        auto *leaf = grid->tree().getFirstNode<0>() + (n >> 9);
        auto *leaf_destiny = destiny->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf->offsetToGlobalCoord(i);
        float new_value = average(coord,grid,n);
        
        leaf_destiny->setValue(coord,new_value);
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}
void copy(nanovdb::FloatGrid* source ,nanovdb::FloatGrid* destiny,u_int64_t leafCount){
    auto kernel = [source,destiny] __device__ (const uint64_t n) {
        auto* leaf_source = source->tree().getFirstNode<0>() + (n >> 9);
        auto* leaf_destiny = destiny->tree().getFirstNode<0>() + (n >> 9);
        const int i = n & 511;
        
        auto coord = leaf_source->offsetToGlobalCoord(i);
        leaf_destiny->setValue(coord,leaf_source->getValue(coord));
    };
    thrust::counting_iterator<uint64_t, thrust::device_system_tag> iter(0);
    thrust::for_each(iter, iter + 512*leafCount, kernel);
}

